#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

#include <assert.h>
#include "hipblas.h"

extern "C" void launch_sgemm_cublas_default(
    float * __restrict__ A, 
    float * __restrict__ B, 
    float * __restrict__ C, 
    const size_t M, 
    const size_t N, 
    const size_t K,
    float alpha)
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float beta = 0;
    hipblasSgemm(
            handle,
            HIPBLAS_OP_N,
            HIPBLAS_OP_N,
            N,              // 
            M,              // 
            K,              // 
            &alpha,         //alpha
            B,              //
            N,              //leading dimension
            A,              //
            K,              //leading dimension
            &beta,          //beta
            C,              //C
            N               //C leading dimension
    );

    hipblasDestroy(handle);
}