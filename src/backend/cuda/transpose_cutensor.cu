#include <stdlib.h>
#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hiptensor.h>

#include <unordered_map>
#include <vector>

// Handle cuTENSOR errors
#define HANDLE_ERROR(x)                                             \
{ const auto err = x;                                               \
    if( err != HIPTENSOR_STATUS_SUCCESS )                              \
    { printf("Error: %s\n", hiptensorGetErrorString(err)); exit(-1); } \
};

#define HANDLE_CUDA_ERROR(x)                                      \
{ const auto err = x;                                             \
    if( err != hipSuccess )                                        \
    { printf("Error: %s\n", hipGetErrorString(err)); exit(-1); } \
};

extern "C" void launch_transpose_cutensor(
  const float* input, 
  float* output, 
  const int shape[4],   // host, 原来的shape
  const int perm[4],    // 采用的perm顺序，比如[0,1,2]->[1,0,2]，则perm={1,0,2,x}
  int rank              // rank
) {
    typedef float floatTypeA;   // input
    typedef float floatTypeC;   // output
    typedef float floatTypeCompute;

    hiptensorDataType_t          const typeA       = HIPTENSOR_R_32F;
    hiptensorDataType_t          const typeC       = HIPTENSOR_R_32F;
    hiptensorComputeDescriptor_t const descCompute = HIPTENSOR_COMPUTE_DESC_32F;

    floatTypeCompute alpha = (floatTypeCompute)1.0f;

    std::vector<int> modeC;
    std::vector<int> modeA;
    for(int i = 0; i < rank; ++i){
      modeA.push_back(i);
      modeC.push_back(perm[i]); 
    }

    int nmodeA = rank;
    int nmodeC = rank;

    std::unordered_map<int, int64_t> extent;
    extent[0] = shape[0];
    extent[1] = shape[1];
    extent[2] = shape[2];
    extent[3] = shape[3];

    std::vector<int64_t> extentA;
    for (auto mode : modeA)
        extentA.push_back(extent[mode]);
    std::vector<int64_t> extentC;
    for (auto mode : modeC)
        extentC.push_back(extent[mode]);

    uint32_t const kAlignment = 128;          // Alignment of the global-memory device pointers (bytes)
    assert(uintptr_t(input) % kAlignment == 0);
    assert(uintptr_t(output) % kAlignment == 0);

    /*************************
     * CUTENSOR
    *************************/

    hiptensorHandle_t handle;
    HANDLE_ERROR(hiptensorCreate(&handle));

    /**********************
     * Create Tensor Descriptors
     **********************/

    hiptensorTensorDescriptor_t  descA;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                                                &descA,
                                                nmodeA,
                                                extentA.data(),
                                                nullptr /* stride */,
                                                typeA,
                                                kAlignment));

    hiptensorTensorDescriptor_t  descC;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                                                &descC,
                                                nmodeC,
                                                extentC.data(),
                                                nullptr /* stride */,
                                                typeC,
                                                kAlignment));

    /*******************************
     * Create Permutation Descriptor
     *******************************/

    hiptensorOperationDescriptor_t  desc;
    HANDLE_ERROR(hiptensorCreatePermutation(handle,
                                          &desc,
                                          descA,
                                          modeA.data(),
                                          HIPTENSOR_OP_IDENTITY,
                                          descC,
                                          modeC.data(),
                                          descCompute));

    /*****************************
     * Optional (but recommended): ensure that the scalar type is correct.
     *****************************/

    hiptensorDataType_t scalarType;
    HANDLE_ERROR(hiptensorOperationDescriptorGetAttribute(handle, desc,
                                                        HIPTENSOR_OPERATION_DESCRIPTOR_SCALAR_TYPE,
                                                        (void*)&scalarType,
                                                        sizeof(scalarType)));

    assert(scalarType == HIPTENSOR_R_32F);

    /**************************
    * Set the algorithm to use
    ***************************/

    const hiptensorAlgo_t algo = HIPTENSOR_ALGO_DEFAULT;

    hiptensorPlanPreference_t  planPref;
    HANDLE_ERROR(hiptensorCreatePlanPreference(handle,
                                              &planPref,
                                              algo,
                                              HIPTENSOR_JIT_MODE_NONE));

    /**************************
     * Create Plan
     **************************/
  
    hiptensorPlan_t  plan;
    HANDLE_ERROR(hiptensorCreatePlan(handle,
                                    &plan,
                                    desc,
                                    planPref,
                                    0 /* workspaceSizeLimit */));

    /**********************
     * Execute
     **********************/
    HANDLE_ERROR(hiptensorPermute(handle,
      plan,
      &alpha, input, output, nullptr /* stream */));
}