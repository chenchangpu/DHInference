#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include <hipcub/hipcub.hpp>
#include <hip/hip_math_constants.h>
#include <assert.h>

#define CUDA_CHECK()  if( (hipPeekAtLastError()) != hipSuccess ) \
  {printf("Error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), \
   __FILE__,__LINE__-1); exit(-1);}

constexpr int kWarpSize = 32;

template<typename T>
struct SumOp {
  __device__ __forceinline__ T operator()(const T& a, const T& b) const { return a + b; }
};

template<typename T>
struct MaxOp {
  __device__ __forceinline__ T operator()(const T& a, const T& b) const { return max(a, b); }
};

template<template<typename> class ReductionOp, typename T, int thread_group_width = kWarpSize>
__inline__ __device__ T WarpAllReduce(T val) {
  for (int mask = thread_group_width / 2; mask > 0; mask /= 2) {
    val = ReductionOp<T>()(val, __shfl_xor_sync(0xffffffff, val, mask, thread_group_width));
  }
  return val;
}

template<template<typename> class ReductionOp, typename T, int block_size>
__inline__ __device__ T BlockAllReduce(T val) {
  typedef hipcub::BlockReduce<T, block_size> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  __shared__ T result_broadcast;
  T result = BlockReduce(temp_storage).Reduce(val, ReductionOp<T>());
  if (threadIdx.x == 0) { result_broadcast = result; }
  __syncthreads();
  return result_broadcast;
}

template<typename T>
__inline__ __device__ T Div(T a, T b);

template<>
__inline__ __device__ float Div<float>(float a, float b) {
#ifdef OF_LAYER_NORM_USE_FAST_MATH
  return __fdividef(a, b);
#else
  return a / b;
#endif
}

template<>
__inline__ __device__ double Div<double>(double a, double b) {
  return a / b;
}

template<typename T>
__inline__ __device__ T Rsqrt(T x);

template<>
__inline__ __device__ float Rsqrt<float>(float x) {
#ifdef OF_LAYER_NORM_USE_FAST_MATH
  return __frsqrt_rn(x);
#else
  return rsqrt(x);
#endif
}

template<>
__inline__ __device__ double Rsqrt<double>(double x) {
  return rsqrt(x);
}

template<class Func>
inline hipError_t GetNumBlocks(Func func, int64_t block_size, size_t dynamic_smem_size,
                                int64_t max_blocks, int64_t waves, int* num_blocks) {
  int dev;
  {
    hipError_t err = hipGetDevice(&dev);
    if (err != hipSuccess) { return err; }
  }
  int sm_count;
  {
    hipError_t err = hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, dev);
    if (err != hipSuccess) { return err; }
  }
  int max_active_blocks;
  {
    hipError_t err = hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks, func,
                                                                    block_size, dynamic_smem_size);
  }
  *num_blocks =
      std::max<int>(1, std::min<int64_t>(max_blocks, sm_count * max_active_blocks * waves));
  return hipSuccess;
}

template<typename T>
struct DefaultComputeType {
  using type = T;
};

template<>
struct DefaultComputeType<half> {
  using type = float;
};

#if CUDA_VERSION >= 11000
template<>
struct DefaultComputeType<hip_bfloat16> {
  using type = float;
};
#endif  // CUDA_VERSION >= 11000

template<typename T>
class HasCanPackAs {
  typedef char one;
  struct two {
    char x[2];
  };

  template<typename C>
  static one test(decltype(&C::CanPackAs));
  template<typename C>
  static two test(...);

 public:
  enum { value = sizeof(test<T>(0)) == sizeof(char) };
};

template<typename T>
typename std::enable_if<HasCanPackAs<T>::value == true, bool>::type CanPackAs(T t,
                                                                              size_t pack_size) {
  return t.CanPackAs(pack_size);
}

template<typename T>
typename std::enable_if<HasCanPackAs<T>::value == false, bool>::type CanPackAs(T t,
                                                                               size_t pack_size) {
  return true;
}

template<typename T, int N>
struct GetPackType {
  using type = typename std::aligned_storage<N * sizeof(T), N * sizeof(T)>::type;
};

template<typename T, int N>
using PackType = typename GetPackType<T, N>::type;

template<typename T, int N>
union Pack {
  static_assert(sizeof(PackType<T, N>) == sizeof(T) * N, "");
  __device__ Pack() {
    // do nothing
  }
  PackType<T, N> storage;
  T elem[N];
};

template<typename SRC, typename DST>
struct DirectLoad {
  using LoadType = DST;
  DirectLoad(const SRC* src, const SRC* gamma, const SRC* beta, int64_t row_size) 
    : src(src), gamma(gamma), beta(beta), row_size(row_size) {}
  template<int N>
  __device__ void load(DST* dst, int64_t row, int64_t col) const {
    Pack<SRC, N> pack;
    const int64_t offset = (row * row_size + col) / N;
    pack.storage = *(reinterpret_cast<const PackType<SRC, N>*>(src) + offset);
#pragma unroll
    for (int i = 0; i < N; ++i) { dst[i] = static_cast<DST>(pack.elem[i]); }
  }
  
  template<int N>
  __device__ void load_gamma(DST* dst, int64_t col) const {
    Pack<SRC, N> pack;
    const int64_t offset = col / N;
    pack.storage = *(reinterpret_cast<const PackType<SRC, N>*>(gamma) + offset);
#pragma unroll
    for (int i = 0; i < N; ++i) { dst[i] = static_cast<DST>(pack.elem[i]); }
  }

  template<int N>
  __device__ void load_beta(DST* dst, int64_t col) const {
    Pack<SRC, N> pack;
    const int64_t offset = col / N;
    pack.storage = *(reinterpret_cast<const PackType<SRC, N>*>(beta) + offset);
#pragma unroll
    for (int i = 0; i < N; ++i) { dst[i] = static_cast<DST>(pack.elem[i]); }
  }

  const SRC* src;
  const SRC* gamma;
  const SRC* beta;
  int64_t row_size;
};

template<typename SRC, typename DST>
struct DirectStore {
  DirectStore(DST* dst, int64_t row_size) : dst(dst), row_size(row_size) {}
  template<int N>
  __device__ void store(const SRC* src, int64_t row, int64_t col) {
    Pack<DST, N> pack;
    const int64_t offset = (row * row_size + col) / N;
#pragma unroll
    for (int i = 0; i < N; ++i) { pack.elem[i] = static_cast<DST>(src[i]); }
    *(reinterpret_cast<PackType<DST, N>*>(dst) + offset) = pack.storage;
  }
  DST* dst;
  int64_t row_size;
};

template<typename T>
inline __device__ void WelfordCombine(T val, T* mean, T* m2, T* count) {
  // Use Welford Online algorithem to compute mean and variance
  // For more details you can refer to:
  // https://en.wikipedia.org/wiki/Algorithms_for_calculating_variance#Welford's_online_algorithm
  *count += 1;
  T delta1 = val - *mean;
  *mean += Div(delta1, *count);
  T delta2 = val - *mean;
  *m2 += delta1 * delta2;
}

template<typename T>
inline __device__ void WelfordCombine(T b_mean, T b_m2, T b_count, T* mean, T* m2, T* count) {
  if (b_count == 0) { return; }
  T new_count = *count + b_count;
  T nb_over_n = Div(b_count, new_count);
  T delta = b_mean - *mean;
  *mean += delta * nb_over_n;
  *m2 += b_m2 + delta * delta * (*count) * nb_over_n;
  *count = new_count;
}

template<typename T, int thread_group_width = kWarpSize>
__inline__ __device__ void WelfordWarpReduce(T thread_mean, T thread_m2, T thread_count, T* mean,
                                             T* m2, T* count) {
  *mean = thread_mean;
  *m2 = thread_m2;
  *count = thread_count;
  for (int mask = thread_group_width / 2; mask > 0; mask /= 2) {
    T b_mean = __shfl_down_sync(0xffffffff, *mean, mask, thread_group_width);
    T b_m2 = __shfl_down_sync(0xffffffff, *m2, mask, thread_group_width);
    T b_count = __shfl_down_sync(0xffffffff, *count, mask, thread_group_width);
    WelfordCombine(b_mean, b_m2, b_count, mean, m2, count);
  }
}

template<typename T, int thread_group_width = kWarpSize>
__inline__ __device__ void WelfordWarpAllReduce(T thread_mean, T thread_m2, T thread_count, T* mean,
                                                T* m2, T* count) {
  WelfordWarpReduce<T, thread_group_width>(thread_mean, thread_m2, thread_count, mean, m2, count);
  *mean = __shfl_sync(0xffffffff, *mean, 0, thread_group_width);
  *m2 = __shfl_sync(0xffffffff, *m2, 0, thread_group_width);
  *count = __shfl_sync(0xffffffff, *count, 0, thread_group_width);
}

template<typename T>
__inline__ __device__ void WelfordBlockAllReduce(T thread_mean, T thread_m2, T thread_count,
                                                 T* result_mean, T* result_m2, T* result_count) {
  __shared__ T mean_shared[kWarpSize];
  __shared__ T m2_shared[kWarpSize];
  __shared__ T count_shared[kWarpSize];
  __shared__ T mean_result_broadcast;
  __shared__ T m2_result_broadcast;
  __shared__ T count_result_broadcast;
  const int lid = threadIdx.x % kWarpSize;
  const int wid = threadIdx.x / kWarpSize;
  T warp_mean = 0;
  T warp_m2 = 0;
  T warp_count = 0;
  WelfordWarpReduce(thread_mean, thread_m2, thread_count, &warp_mean, &warp_m2, &warp_count);
  __syncthreads();
  if (lid == 0) {
    mean_shared[wid] = warp_mean;
    m2_shared[wid] = warp_m2;
    count_shared[wid] = warp_count;
  }
  __syncthreads();
  if (wid == 0) {
    if (threadIdx.x < blockDim.x / kWarpSize) {
      warp_mean = mean_shared[lid];
      warp_m2 = m2_shared[lid];
      warp_count = count_shared[lid];
    } else {
      warp_mean = static_cast<T>(0);
      warp_m2 = static_cast<T>(0);
      warp_count = static_cast<T>(0);
    }
    __syncwarp();
    T block_mean = 0;
    T block_m2 = 0;
    T block_count = 0;
    WelfordWarpReduce(warp_mean, warp_m2, warp_count, &block_mean, &block_m2, &block_count);
    if (lid == 0) {
      mean_result_broadcast = block_mean;
      m2_result_broadcast = block_m2;
      count_result_broadcast = block_count;
    }
  }
  __syncthreads();
  *result_mean = mean_result_broadcast;
  *result_m2 = m2_result_broadcast;
  *result_count = count_result_broadcast;
}

template<typename LOAD, typename STORE, typename ComputeType, int pack_size,
         int max_cols_per_thread, int min_cols_per_thread, int thread_group_width,
         int rows_per_access, bool padding>
__global__ void LayerNormWarpImpl(LOAD load, STORE store, const int64_t rows, const int64_t cols,
                                  const double epsilon) {
  using LoadType = typename LOAD::LoadType;
  static_assert(max_cols_per_thread % pack_size == 0, "");
  static_assert(min_cols_per_thread % pack_size == 0, "");
  static_assert(thread_group_width <= kWarpSize, "");
  static_assert(kWarpSize % thread_group_width == 0, "");
  constexpr int max_num_packs = max_cols_per_thread / pack_size;
  constexpr int min_num_packs = min_cols_per_thread / pack_size;
  assert(cols <= max_cols_per_thread * thread_group_width);
  ComputeType buf[rows_per_access][max_cols_per_thread];
  ComputeType gamma_buf[max_cols_per_thread];
  ComputeType beta_buf[max_cols_per_thread];
  const int64_t global_thread_group_id = blockIdx.x * blockDim.y + threadIdx.y;
  const int64_t num_global_thread_group = gridDim.x * blockDim.y;
  const int64_t lane_id = threadIdx.x;
  const int64_t step = num_global_thread_group * rows_per_access;
  
  // 预加载gamma和beta
#pragma unroll
  for (int pack_id = 0; pack_id < min_num_packs; ++pack_id) {
    const int col = (pack_id * thread_group_width + lane_id) * pack_size;
    const int pack_offset = pack_id * pack_size;
    LoadType pack[pack_size];
    load.template load_gamma<pack_size>(pack, col);
#pragma unroll
    for (int i = 0; i < pack_size; ++i) {
      gamma_buf[pack_offset + i] = static_cast<ComputeType>(pack[i]);
    }
    load.template load_beta<pack_size>(pack, col);
#pragma unroll
    for (int i = 0; i < pack_size; ++i) {
      beta_buf[pack_offset + i] = static_cast<ComputeType>(pack[i]);
    }
  }

  for (int64_t row = global_thread_group_id * rows_per_access; row < rows; row += step) {
    ComputeType thread_mean[rows_per_access];
    ComputeType thread_m2[rows_per_access];
    ComputeType thread_count[rows_per_access];
#pragma unroll
    for (int row_id = 0; row_id < rows_per_access; ++row_id) {
      thread_mean[row_id] = 0;
      thread_m2[row_id] = 0;
      thread_count[row_id] = 0;
      ComputeType* row_buf = buf[row_id];
#pragma unroll
      for (int pack_id = 0; pack_id < min_num_packs; ++pack_id) {
        const int col = (pack_id * thread_group_width + lane_id) * pack_size;
        const int pack_offset = pack_id * pack_size;
        LoadType pack[pack_size];
        load.template load<pack_size>(pack, row + row_id, col);
#pragma unroll
        for (int i = 0; i < pack_size; ++i) {
          row_buf[pack_offset + i] = static_cast<ComputeType>(pack[i]);
          WelfordCombine(row_buf[pack_offset + i], thread_mean + row_id, thread_m2 + row_id,
                         thread_count + row_id);
        }
      }
      for (int pack_id = min_num_packs; pack_id < max_num_packs; ++pack_id) {
        const int col = (pack_id * thread_group_width + lane_id) * pack_size;
        const int pack_offset = pack_id * pack_size;
        if (!padding || col < cols) {
          LoadType pack[pack_size];
          load.template load<pack_size>(pack, row + row_id, col);
#pragma unroll
          for (int i = 0; i < pack_size; ++i) {
            row_buf[pack_offset + i] = static_cast<ComputeType>(pack[i]);
            WelfordCombine(row_buf[pack_offset + i], thread_mean + row_id, thread_m2 + row_id,
                           thread_count + row_id);
          }
        } else {
#pragma unroll
          for (int i = 0; i < pack_size; ++i) { row_buf[pack_offset + i] = 0; }
        }
      }
    }
    ComputeType warp_mean[rows_per_access];
    ComputeType warp_m2[rows_per_access];
    ComputeType warp_count[rows_per_access];
#pragma unroll
    for (int row_id = 0; row_id < rows_per_access; ++row_id) {
      int global_row_id = row + row_id;
      ComputeType* row_buf = buf[row_id];
      WelfordWarpAllReduce<ComputeType, thread_group_width>(
          thread_mean[row_id], thread_m2[row_id], thread_count[row_id], warp_mean + row_id,
          warp_m2 + row_id, warp_count + row_id);
      ComputeType row_mean = warp_mean[row_id];
      ComputeType row_variance =
          max(Div(warp_m2[row_id], warp_count[row_id]), static_cast<ComputeType>(0.0));
      ComputeType row_inv_var = Rsqrt(row_variance + static_cast<ComputeType>(epsilon));

#pragma unroll
      for (int i = 0; i < max_cols_per_thread; ++i) {
        row_buf[i] = (row_buf[i] - row_mean) * row_inv_var * gamma_buf[i] + beta_buf[i];
      }
#pragma unroll
      for (int i = 0; i < min_num_packs; ++i) {
        const int col = (i * thread_group_width + lane_id) * pack_size;
        store.template store<pack_size>(row_buf + i * pack_size, global_row_id, col);
      }
#pragma unroll
      for (int i = min_num_packs; i < max_num_packs; ++i) {
        const int col = (i * thread_group_width + lane_id) * pack_size;
        if (!padding || col < cols) {
          store.template store<pack_size>(row_buf + i * pack_size, global_row_id, col);
        }
      }
    }
  }
}

template<typename LOAD, typename STORE, typename ComputeType, int pack_size,
         int max_cols_per_thread, int min_cols_per_thread, int thread_group_width,
         int rows_per_access, bool padding>
inline hipError_t LaunchLayerNormWarpImpl(hipStream_t stream, LOAD load, STORE store,
                                           const int64_t rows, const int64_t cols,
                                           const double epsilon) {
  constexpr int block_size = 128;
  constexpr int waves = 32;
  static_assert(block_size % thread_group_width == 0, "");
  constexpr int thread_groups_per_block = block_size / thread_group_width;
  dim3 block_dim(thread_group_width, thread_groups_per_block);
  const int64_t num_blocks =
      (rows / rows_per_access + thread_groups_per_block - 1) / thread_groups_per_block;
  int grid_dim_x;
  {
    hipError_t err = GetNumBlocks(
        LayerNormWarpImpl<LOAD, STORE, ComputeType, pack_size, max_cols_per_thread,
                          min_cols_per_thread, thread_group_width, rows_per_access, padding>,
        block_size, 0, num_blocks, waves, &grid_dim_x);
    if (err != hipSuccess) { return err; }
  }
  LayerNormWarpImpl<LOAD, STORE, ComputeType, pack_size, max_cols_per_thread, min_cols_per_thread,
                    thread_group_width, rows_per_access, padding>
      <<<grid_dim_x, block_dim, 0, stream>>>(load, store, rows, cols, epsilon);
  return hipPeekAtLastError();
}

template<typename LOAD, typename STORE, typename ComputeType, int pack_size,
         int max_cols_per_thread, int min_cols_per_thread, int thread_group_width,
         int rows_per_access>
inline hipError_t DispatchLayerNormWarpImplPadding(hipStream_t stream, LOAD load, STORE store,
                                                    const int64_t rows, const int64_t cols,
                                                    const double epsilon) {
  if (cols == max_cols_per_thread * thread_group_width) {
    // when not padding, min_cols_per_thread must equals to max_cols_per_thread, pass
    // max_cols_per_thread as min_cols_per_thread and max_cols_per_thread param.
    return LaunchLayerNormWarpImpl<LOAD, STORE, ComputeType, pack_size, max_cols_per_thread,
                                   max_cols_per_thread, thread_group_width, rows_per_access, false>(
        stream, load, store, rows, cols, epsilon);
  } else {
    return LaunchLayerNormWarpImpl<LOAD, STORE, ComputeType, pack_size, max_cols_per_thread,
                                   min_cols_per_thread, thread_group_width, rows_per_access, true>(
        stream, load, store, rows, cols, epsilon);
  }
}

template<typename LOAD, typename STORE, typename ComputeType, int pack_size>
typename std::enable_if<pack_size == 1, hipError_t>::type DispatchLayerNormWarpImplCols(
    hipStream_t stream, LOAD load, STORE store, const int64_t rows, const int64_t cols,
    const double epsilon) {
  if (cols <= 0) { return hipErrorInvalidValue; }
#define DEFINE_ONE_ELIF(thread_group_width)                                                      \
  else if (cols <= (thread_group_width)*pack_size) {                                             \
    if (rows % 2 == 0) {                                                                         \
      return DispatchLayerNormWarpImplPadding<LOAD, STORE, ComputeType, pack_size, pack_size, 0, \
                                              thread_group_width, 2>(                            \
          stream, load, store, rows, cols, epsilon);                         \
    } else {                                                                                     \
      return DispatchLayerNormWarpImplPadding<LOAD, STORE, ComputeType, pack_size, pack_size, 0, \
                                              thread_group_width, 1>(                            \
          stream, load, store, rows, cols, epsilon);                         \
    }                                                                                            \
  }
  DEFINE_ONE_ELIF(4)
  DEFINE_ONE_ELIF(8)
  DEFINE_ONE_ELIF(16)
  DEFINE_ONE_ELIF(32)
#undef DEFINE_ONE_ELIF
#define DEFINE_ONE_ELIF(max_col, min_col)                                                          \
  else if (cols <= (max_col)*kWarpSize) {                                                          \
    return DispatchLayerNormWarpImplPadding<LOAD, STORE, ComputeType, pack_size, max_col, min_col, \
                                            kWarpSize, 1>(stream, load, store, rows, cols,         \
                                                          epsilon);            \
  }
  DEFINE_ONE_ELIF(2, 1)
  DEFINE_ONE_ELIF(4, 2)
  DEFINE_ONE_ELIF(8, 4)
  DEFINE_ONE_ELIF(12, 8)
  DEFINE_ONE_ELIF(16, 12)
  DEFINE_ONE_ELIF(20, 16)
  DEFINE_ONE_ELIF(24, 20)
  DEFINE_ONE_ELIF(28, 24)
  DEFINE_ONE_ELIF(32, 28)
#undef DEFINE_ONE_ELIF
  else {
    return hipErrorInvalidValue;
  }
}

template<typename LOAD, typename STORE, typename ComputeType, int pack_size>
typename std::enable_if<pack_size == 2, hipError_t>::type DispatchLayerNormWarpImplCols(
    hipStream_t stream, LOAD load, STORE store, const int64_t rows, const int64_t cols,
    const double epsilon) {
  if (cols <= 0) { return hipErrorInvalidValue; }
#define DEFINE_ONE_ELIF(thread_group_width)                                                      \
  else if (cols <= (thread_group_width)*pack_size) {                                             \
    if (rows % 2 == 0) {                                                                         \
      return DispatchLayerNormWarpImplPadding<LOAD, STORE, ComputeType, pack_size, pack_size, 0, \
                                              thread_group_width, 2>(                            \
          stream, load, store, rows, cols, epsilon);                         \
    } else {                                                                                     \
      return DispatchLayerNormWarpImplPadding<LOAD, STORE, ComputeType, pack_size, pack_size, 0, \
                                              thread_group_width, 1>(                            \
          stream, load, store, rows, cols, epsilon);                         \
    }                                                                                            \
  }
  DEFINE_ONE_ELIF(4)
  DEFINE_ONE_ELIF(8)
  DEFINE_ONE_ELIF(16)
  DEFINE_ONE_ELIF(32)
#undef DEFINE_ONE_ELIF
#define DEFINE_ONE_ELIF(max_col, min_col)                                                          \
  else if ((cols <= (max_col)*kWarpSize) && (cols > (min_col)*kWarpSize)) {                        \
    return DispatchLayerNormWarpImplPadding<LOAD, STORE, ComputeType, pack_size, max_col, min_col, \
                                            kWarpSize, 1>(stream, load, store, rows, cols,         \
                                                          epsilon);            \
  }
  DEFINE_ONE_ELIF(4, 2)
  DEFINE_ONE_ELIF(8, 4)
  DEFINE_ONE_ELIF(12, 8)
  DEFINE_ONE_ELIF(16, 12)
  DEFINE_ONE_ELIF(20, 16)
  DEFINE_ONE_ELIF(24, 20)
  DEFINE_ONE_ELIF(28, 24)
  DEFINE_ONE_ELIF(32, 28)
#undef DEFINE_ONE_ELIF
  else {
    return hipErrorInvalidValue;
  }
}

template<typename LOAD, typename STORE, typename ComputeType>
struct DispatchLayerNormWarpImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                         const int64_t cols, const double epsilon) {
    if (cols % 2 == 0 && CanPackAs<LOAD>(load, 2) && CanPackAs<STORE>(store, 2)) {
      return DispatchLayerNormWarpImplCols<LOAD, STORE, ComputeType, 2>(
          stream, load, store, rows, cols, epsilon);
    } else {
      return DispatchLayerNormWarpImplCols<LOAD, STORE, ComputeType, 1>(
          stream, load, store, rows, cols, epsilon);
    }
  }
};

template<typename LOAD, typename STORE, typename ComputeType>
inline hipError_t DispatchLayerNormWarpImpl(hipStream_t stream, LOAD load, STORE store,
                                             const int64_t rows, const int64_t cols,
                                             const double epsilon) {
  return DispatchLayerNormWarpImplPackSize<LOAD, STORE, ComputeType>()(
      stream, load, store, rows, cols, epsilon);
}

template<typename LOAD, typename STORE, typename ComputeType, int pack_size, int block_size>
__global__ void LayerNormBlockSMemImpl(LOAD load, STORE store, const int64_t rows,
                                       const int64_t cols, const double epsilon) {
  using LoadType = typename LOAD::LoadType;
  extern __shared__ __align__(sizeof(double)) unsigned char shared_buf[];
  auto* buf = reinterpret_cast<LoadType*>(shared_buf);
  const int tid = threadIdx.x;
  assert(cols % pack_size == 0);
  const int num_packs = static_cast<int>(cols) / pack_size;
  
  // 在共享内存中为gamma和beta分配空间
  LoadType* gamma_buf = buf + cols;
  LoadType* beta_buf = gamma_buf + cols;
  
  // 预加载gamma和beta到共享内存
  for (int pack_id = tid; pack_id < num_packs; pack_id += block_size) {
    LoadType pack[pack_size];
    load.template load_gamma<pack_size>(pack, pack_id * pack_size);
#pragma unroll
    for (int i = 0; i < pack_size; ++i) {
      gamma_buf[pack_id * pack_size + i] = pack[i];
    }
    load.template load_beta<pack_size>(pack, pack_id * pack_size);
#pragma unroll
    for (int i = 0; i < pack_size; ++i) {
      beta_buf[pack_id * pack_size + i] = pack[i];
    }
  }
  __syncthreads();
  
  for (int64_t row = blockIdx.x; row < rows; row += gridDim.x) {
    ComputeType thread_mean = 0;
    ComputeType thread_m2 = 0;
    ComputeType thread_count = 0;
    for (int pack_id = tid; pack_id < num_packs; pack_id += block_size) {
      LoadType pack[pack_size];
      load.template load<pack_size>(pack, row, pack_id * pack_size);
#pragma unroll
      for (int i = 0; i < pack_size; ++i) {
        buf[i * num_packs + pack_id] = pack[i];
        WelfordCombine(static_cast<ComputeType>(pack[i]), &thread_mean, &thread_m2, &thread_count);
      }
    }
    ComputeType row_mean = 0;
    ComputeType row_m2 = 0;
    ComputeType row_count = 0;
    WelfordBlockAllReduce<ComputeType>(thread_mean, thread_m2, thread_count, &row_mean, &row_m2,
                                       &row_count);
    ComputeType row_variance = max(Div(row_m2, row_count), static_cast<ComputeType>(0.0));
    ComputeType row_inv_var = Rsqrt(row_variance + static_cast<ComputeType>(epsilon));

    for (int pack_id = tid; pack_id < num_packs; pack_id += block_size) {
      ComputeType pack[pack_size];
#pragma unroll
      for (int i = 0; i < pack_size; ++i) {
        const int col_idx = pack_id * pack_size + i;
        pack[i] = (static_cast<ComputeType>(buf[i * num_packs + pack_id]) - row_mean) * row_inv_var 
                  * static_cast<ComputeType>(gamma_buf[col_idx]) 
                  + static_cast<ComputeType>(beta_buf[col_idx]);
      }
      store.template store<pack_size>(pack, row, pack_id * pack_size);
    }
  }
}

template<typename LOAD, typename STORE, typename ComputeType, int pack_size, int block_size>
inline hipError_t LaunchLayerNormBlockSMemImpl(hipStream_t stream, LOAD load, STORE store,
                                                int smem, const int64_t rows, const int64_t cols,
                                                const double epsilon) {
  constexpr int waves = 32;
  int grid_dim_x;
  {
    hipError_t err =
        GetNumBlocks(LayerNormBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size>,
                     block_size, smem, rows, waves, &grid_dim_x);
    if (err != hipSuccess) { return err; }
  }
  LayerNormBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size>
      <<<grid_dim_x, block_size, smem, stream>>>(load, store, rows, cols, epsilon);
  return hipPeekAtLastError();
}

template<typename Func>
hipError_t MaximizeDynamicSharedMemorySize(Func func, const int max_smem_size) {
  hipFuncAttributes attr{};
  hipError_t err = hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(func));
  if (err != hipSuccess) { return err; }
  constexpr int reserved_smem = 1024;  // 1K
  return hipFuncSetAttribute(reinterpret_cast<const void*>(func), hipFuncAttributeMaxDynamicSharedMemorySize,
                              max_smem_size - attr.sharedSizeBytes - reserved_smem);
}

template<typename LOAD, typename STORE, typename ComputeType, int pack_size>
inline hipError_t TryDispatchLayerNormBlockSMemImplBlockSize(
    hipStream_t stream, LOAD load, STORE store, const int64_t rows, const int64_t cols,
    const double epsilon, bool* success) {
  constexpr int block_size_conf_1 = 128;
  constexpr int block_size_conf_2 = 256;
  constexpr int block_size_conf_3 = 512;
  constexpr int block_size_conf_4 = 1024;

  int dev = 0;
  {
    hipError_t err = hipGetDevice(&dev);
    if (err != hipSuccess) { return err; }
  }

  int sm_count = 0;
  {
    hipError_t err = hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, dev);
    if (err != hipSuccess) { return err; }
  }

  static const bool max_smem_configed = [=]() {
    int max_smem_size = 0;
    hipError_t err =
        hipDeviceGetAttribute(&max_smem_size, hipDeviceAttributeSharedMemPerBlockOptin, dev);
    if (err != hipSuccess) { return false; }

    err = MaximizeDynamicSharedMemorySize(
        LayerNormBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_1>,
        max_smem_size);
    if (err != hipSuccess) { return false; }
    err = MaximizeDynamicSharedMemorySize(
        LayerNormBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_2>,
        max_smem_size);
    if (err != hipSuccess) { return false; }
    err = MaximizeDynamicSharedMemorySize(
        LayerNormBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_3>,
        max_smem_size);
    if (err != hipSuccess) { return false; }
    err = MaximizeDynamicSharedMemorySize(
        LayerNormBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_4>,
        max_smem_size);
    if (err != hipSuccess) { return false; }

    return true;
  }();

  // 更新shared memory大小: 原始数据 + gamma + beta
  const size_t smem = cols * sizeof(typename LOAD::LoadType) * 3;

  int max_active_blocks_conf_1;
  {
    hipError_t err = hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks_conf_1,
        LayerNormBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_1>,
        block_size_conf_1, smem);
    if (err != hipSuccess) { return err; }
  }
  if (max_active_blocks_conf_1 <= 0) {
    *success = false;
    return hipSuccess;
  }

  int max_active_blocks_conf_4;
  {
    hipError_t err = hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks_conf_4,
        LayerNormBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_4>,
        block_size_conf_4, smem);
    if (err != hipSuccess) { return err; }
  }

  if (max_active_blocks_conf_4 == max_active_blocks_conf_1
      || (max_active_blocks_conf_4 > 0 && rows <= sm_count)) {
    *success = true;
    return LaunchLayerNormBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_4>(
        stream, load, store, smem, rows, cols, epsilon);
  }

  int max_active_blocks_conf_3;
  {
    hipError_t err = hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks_conf_3,
        LayerNormBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_3>,
        block_size_conf_3, smem);
    if (err != hipSuccess) { return err; }
  }
  if (max_active_blocks_conf_3 == max_active_blocks_conf_1
      || (max_active_blocks_conf_3 > 0 && rows <= sm_count)) {
    *success = true;
    return LaunchLayerNormBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_3>(
        stream, load, store, smem, rows, cols, epsilon);
  }

  int max_active_blocks_conf_2;
  {
    hipError_t err = hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks_conf_2,
        LayerNormBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_2>,
        block_size_conf_2, smem);
    if (err != hipSuccess) { return err; }
  }
  if (max_active_blocks_conf_2 == max_active_blocks_conf_1
      || (max_active_blocks_conf_2 > 0 && rows <= sm_count)) {
    *success = true;
    return LaunchLayerNormBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_2>(
        stream, load, store, smem, rows, cols, epsilon);
  }

  *success = true;
  return LaunchLayerNormBlockSMemImpl<LOAD, STORE, ComputeType, pack_size, block_size_conf_1>(
      stream, load, store, smem, rows, cols, epsilon);
}

template<typename LOAD, typename STORE, typename ComputeType>
struct TryDispatchLayerNormBlockSMemImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                         const int64_t cols, const double epsilon, bool* success) {
    if (cols % 4 == 0 && CanPackAs<LOAD>(load, 4) && CanPackAs<STORE>(store, 4)) {
      return TryDispatchLayerNormBlockSMemImplBlockSize<LOAD, STORE, ComputeType, 4>(
          stream, load, store, rows, cols, epsilon, success);
    } else if (cols % 2 == 0 && CanPackAs<LOAD>(load, 2) && CanPackAs<STORE>(store, 2)) {
      return TryDispatchLayerNormBlockSMemImplBlockSize<LOAD, STORE, ComputeType, 2>(
          stream, load, store, rows, cols, epsilon, success);
    } else {
      return TryDispatchLayerNormBlockSMemImplBlockSize<LOAD, STORE, ComputeType, 1>(
          stream, load, store, rows, cols, epsilon, success);
    }
  }
};

template<typename LOAD, typename STORE, typename ComputeType>
inline hipError_t TryDispatchLayerNormBlockSMemImpl(hipStream_t stream, LOAD load, STORE store,
                                                     const int64_t rows, const int64_t cols,
                                                     const double epsilon, bool* success) {
  return TryDispatchLayerNormBlockSMemImplPackSize<LOAD, STORE, ComputeType>()(
      stream, load, store, rows, cols, epsilon, success);
}

template<typename LOAD, typename STORE, typename ComputeType, int pack_size, int block_size>
__global__ void __launch_bounds__(1024)
    LayerNormBlockUncachedImpl(LOAD load, STORE store, const int64_t rows, const int64_t cols,
                               const double epsilon) {
  using LoadType = typename LOAD::LoadType;
  const int tid = threadIdx.x;
  assert(cols % pack_size == 0);
  const int num_packs = static_cast<int>(cols) / pack_size;
  
  // 预加载gamma和beta
  LoadType gamma_buf[pack_size];
  LoadType beta_buf[pack_size];
  
  for (int64_t row = blockIdx.x; row < rows; row += gridDim.x) {
    ComputeType thread_mean = 0;
    ComputeType thread_m2 = 0;
    ComputeType thread_count = 0;
    for (int pack_id = tid; pack_id < num_packs; pack_id += block_size) {
      LoadType pack[pack_size];
      load.template load<pack_size>(pack, row, pack_id * pack_size);
#pragma unroll
      for (int i = 0; i < pack_size; ++i) {
        WelfordCombine(static_cast<ComputeType>(pack[i]), &thread_mean, &thread_m2, &thread_count);
      }
    }
    ComputeType row_mean = 0;
    ComputeType row_m2 = 0;
    ComputeType row_count = 0;
    WelfordBlockAllReduce<ComputeType>(thread_mean, thread_m2, thread_count, &row_mean, &row_m2,
                                       &row_count);
    ComputeType row_variance = max(Div(row_m2, row_count), static_cast<ComputeType>(0.0));
    ComputeType row_inv_var = Rsqrt(row_variance + static_cast<ComputeType>(epsilon));

    for (int pack_id = tid; pack_id < num_packs; pack_id += block_size) {
      LoadType pack[pack_size];
      ComputeType dst_pack[pack_size];
      const int pack_offset = pack_id * pack_size;
      
      // 加载输入数据
      load.template load<pack_size>(pack, row, pack_offset);
      // 加载gamma和beta
      load.template load_gamma<pack_size>(gamma_buf, pack_offset);
      load.template load_beta<pack_size>(beta_buf, pack_offset);
      
#pragma unroll
      for (int i = 0; i < pack_size; ++i) {
        dst_pack[i] = (static_cast<ComputeType>(pack[i]) - row_mean) * row_inv_var
                      * static_cast<ComputeType>(gamma_buf[i])
                      + static_cast<ComputeType>(beta_buf[i]);
      }
      store.template store<pack_size>(dst_pack, row, pack_offset);
    }
  }
}

template<typename LOAD, typename STORE, typename ComputeType, int pack_size>
inline hipError_t LaunchLayerNormBlockUncachedImpl(hipStream_t stream, LOAD load, STORE store,
                                                    const int64_t rows, const int64_t cols,
                                                    const double epsilon) {
  constexpr int block_size = 1024;
  constexpr int waves = 32;
  int grid_dim_x;
  {
    hipError_t err =
        GetNumBlocks(LayerNormBlockUncachedImpl<LOAD, STORE, ComputeType, pack_size, block_size>,
                     block_size, 0, rows, waves, &grid_dim_x);
    if (err != hipSuccess) { return err; }
  }
  LayerNormBlockUncachedImpl<LOAD, STORE, ComputeType, pack_size, block_size>
      <<<grid_dim_x, block_size, 0, stream>>>(load, store, rows, cols, epsilon);
  return hipPeekAtLastError();
}

template<typename LOAD, typename STORE, typename ComputeType>
struct DispatchLayerNormBlockUncachedImplPackSize {
  hipError_t operator()(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                         const int64_t cols, const double epsilon) {
    if (cols % 4 == 0 && CanPackAs<LOAD>(load, 4) && CanPackAs<STORE>(store, 4)) {
      return LaunchLayerNormBlockUncachedImpl<LOAD, STORE, ComputeType, 4>(
          stream, load, store, rows, cols, epsilon);
    } else if (cols % 2 == 0 && CanPackAs<LOAD>(load, 2) && CanPackAs<STORE>(store, 2)) {
      return LaunchLayerNormBlockUncachedImpl<LOAD, STORE, ComputeType, 2>(
          stream, load, store, rows, cols, epsilon);
    } else {
      return LaunchLayerNormBlockUncachedImpl<LOAD, STORE, ComputeType, 1>(
          stream, load, store, rows, cols, epsilon);
    }
  }
};

template<typename LOAD, typename STORE, typename ComputeType>
inline hipError_t DispatchLayerNormBlockUncachedImpl(hipStream_t stream, LOAD load, STORE store,
                                                      const int64_t rows, const int64_t cols,
                                                      const double epsilon) {
  return DispatchLayerNormBlockUncachedImplPackSize<LOAD, STORE, ComputeType>()(
      stream, load, store, rows, cols, epsilon);
}

template<typename LOAD, typename STORE, typename ComputeType>
inline typename std::enable_if<!std::is_same<ComputeType, double>::value, hipError_t>::type
DispatchLayerNorm(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                  const int64_t cols, const double epsilon) {
  if (cols <= 1024) {
    return DispatchLayerNormWarpImpl<LOAD, STORE, ComputeType>(stream, load, store, rows, cols,
                                                               epsilon);
  } else {
    bool dispatch_smem_impl_success;
    {
      hipError_t err = TryDispatchLayerNormBlockSMemImpl<LOAD, STORE, ComputeType>(
          stream, load, store, rows, cols, epsilon,
          &dispatch_smem_impl_success);
      if (err != hipSuccess) { return err; }
    }
    if (!dispatch_smem_impl_success) {
      return DispatchLayerNormBlockUncachedImpl<LOAD, STORE, ComputeType>(
          stream, load, store, rows, cols, epsilon);
    }
    return hipSuccess;
  }
}

template<typename LOAD, typename STORE, typename ComputeType>
inline typename std::enable_if<std::is_same<ComputeType, double>::value, hipError_t>::type
DispatchLayerNorm(hipStream_t stream, LOAD load, STORE store, const int64_t rows,
                  const int64_t cols, const double epsilon) {
  return DispatchLayerNormBlockUncachedImpl<LOAD, STORE, ComputeType>(
      stream, load, store, rows, cols, epsilon);
}

extern "C" void launch_layernorm_oneflow(float* d_x, float* d_y, float* g, float* b, int N, int K) {
    // N是batch_size，K是hidden_dim
    using ComputeType = typename DefaultComputeType<float>::type;
    
    // 创建输入和输出的加载器
    DirectLoad<float, ComputeType> load(d_x, g, b, K);
    DirectStore<ComputeType, float> store(d_y, K);
    
    // 创建cuda stream
    hipStream_t stream;
    hipStreamCreate(&stream);
    
    // 调用LayerNorm实现
    const double eps = 1e-5;
    DispatchLayerNorm<decltype(load), decltype(store), ComputeType>(
        stream, load, store, N, K, eps);
    
    // 同步stream并销毁
    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);
}

// int main(){
//   const int rows = 1024;
//   const int cols = 1024;
//   const int N = rows * cols;
//   using ComputeType = typename DefaultComputeType<float>::type;
  
//   // 分配并初始化输入数据
//   float* input_host = (float*)malloc(N*sizeof(float));
//   float *input_device;
//   hipMalloc((void **)&input_device, N*sizeof(float));
//   for (int i = 0; i < N; i++) input_host[i] = 1.0;
//   hipMemcpy(input_device, input_host, N*sizeof(float), hipMemcpyHostToDevice);
  
//   // 分配并初始化gamma
//   float* gamma_host = (float*)malloc(cols*sizeof(float));
//   float *gamma_device;
//   hipMalloc((void **)&gamma_device, cols*sizeof(float));
//   for (int i = 0; i < cols; i++) gamma_host[i] = 1.0;  // gamma初始化为1
//   hipMemcpy(gamma_device, gamma_host, cols*sizeof(float), hipMemcpyHostToDevice);
  
//   // 分配并初始化beta
//   float* beta_host = (float*)malloc(cols*sizeof(float));
//   float *beta_device;
//   hipMalloc((void **)&beta_device, cols*sizeof(float));
//   for (int i = 0; i < cols; i++) beta_host[i] = 0.0;  // beta初始化为0
//   hipMemcpy(beta_device, beta_host, cols*sizeof(float), hipMemcpyHostToDevice);
  
//   // 创建DirectLoad实例，现在需要传入gamma和beta
//   DirectLoad<float, ComputeType> load(input_device, gamma_device, beta_device, cols);
  
//   // 输出相关
//   float *output_host = (float*)malloc(N * sizeof(float));
//   float *output_device;
//   hipMalloc((void **)&output_device, N * sizeof(float));
//   DirectStore<ComputeType, float> store(output_device, cols);
  
//   float eps = 1e-5;
  
//   hipStream_t stream;
//   hipStreamCreate(&stream);
//   DispatchLayerNorm<decltype(load), decltype(store), ComputeType>(
//         stream, load, store, rows, cols, eps);
//   CUDA_CHECK();
  
//   hipMemcpy(output_host, output_device, N * sizeof(float), hipMemcpyDeviceToHost);
  
//   // 打印结果
//   printf("LayerNorm结果 (前32个元素):\n");
//   for (int i = 0; i < 32; i++){
//     printf("%.5f\n", output_host[i]);
//   }
  
//   // 释放内存
//   hipFree(input_device);
//   hipFree(gamma_device);
//   hipFree(beta_device);
//   hipFree(output_device);
//   free(input_host);
//   free(gamma_host);
//   free(beta_host);
//   free(output_host);
  
//   return 0;
// }