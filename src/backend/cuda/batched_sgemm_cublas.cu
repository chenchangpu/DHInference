#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

#include <assert.h>
#include "hipblas.h"

extern "C" void launch_batched_sgemm_cublas_default(
    float * __restrict__ A,     // [Batch, M, K]
    float * __restrict__ B,     // [Batch, K, N]
    float * __restrict__ C,     // [Batch, M, N]
    const size_t Batch,
    const size_t M, 
    const size_t N, 
    const size_t K,
    float alpha)
{
    // host
    const float** Aarray = (const float**)malloc(Batch * sizeof(float*));
    const float** Barray = (const float**)malloc(Batch * sizeof(float*));
    float** Carray = (float**)malloc(Batch * sizeof(float*));
    // device
    const float** d_Aarray;
    const float** d_Barray;
    float** d_Carray;
    hipMalloc(&d_Aarray, Batch * sizeof(float*));
    hipMalloc(&d_Barray, Batch * sizeof(float*));
    hipMalloc(&d_Carray, Batch * sizeof(float*));
    // 指针数组赋值
    for(int i = 0; i < Batch; ++i){
        Aarray[i] = (const float*)(A + i * M * K);
        Barray[i] = (const float*)(B + i * K * N);
        Carray[i] = (float*)(C + i * M * N);
    }
    hipMemcpy(d_Aarray, Aarray, Batch * sizeof(float*), hipMemcpyHostToDevice);
    hipMemcpy(d_Barray, Barray, Batch * sizeof(float*), hipMemcpyHostToDevice);
    hipMemcpy(d_Carray, Carray, Batch * sizeof(float*), hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float beta = 0;
    hipblasSgemmBatched(
            handle,
            HIPBLAS_OP_N,
            HIPBLAS_OP_N,
            N, M, K,              
            &alpha, 
            d_Barray, N,
            d_Aarray, K, 
            &beta,
            d_Carray, N,
            Batch
    );

    hipblasDestroy(handle);
}